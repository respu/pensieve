#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>

#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

#include <chrono>
#include <algorithm>
#include <numeric>
#include <cmath>

template<typename Container>
cv::Mat copy_to_Mat(Container& container, int rows, int cols) {
    cv::Mat img(rows, cols, CV_32FC3);

    for(int i = 0; i != img.rows; ++i) {
        for(int j = 0; j != img.cols * 3; ++j) {
            img.at<float>(i,j) = container[img.cols * 3 * i + j];
        }
    }

    return img;
}

// Dear opencv why don't you do this for me?
cv::Mat transform_to_uchar(cv::Mat& img) {
    cv::Mat transformed(img.rows, img.cols, CV_8UC3);
    for(auto i = 0; i != img.rows; ++i) {
        for(auto j = 0; j != img.cols * 3; ++j) {
            transformed.at<unsigned char>(i, j) = img.at<float>(i, j) * 255.f;
        }
    }

    return img;
}

template<typename T>
__host__ __device__
void apply_mandel_brot(T* dest, int x, int y, int rows, int cols, const int iterations) {
    // calculate step
	const float x_scaler = 3.5f / static_cast<float>(cols * 3.f);
    const float y_scaler = 3.0f / static_cast<float>(rows);
    
    const float x0 = x * x_scaler - 2.5f;
	const float y0 = y * y_scaler - 1.5f;

	float real = 0.f;
	float img = 0.f;

	int iter = 0;
	while(real * real + img * img < 4.f && iter < iterations) {
        float real_temp = real * real - img * img + x0;
        img = 2.f * real * img + y0;
        real = real_temp;

        ++iter;
    }

    // coloring step
    const int base_index = cols * 3 * y + x;
    if(iter == iterations) {
        dest[base_index] = 0.f;
        dest[base_index + 1] = 0.f;
        dest[base_index + 2] = 0.f;
    }
    else {
        // random formula from the www; probably very slow
        const float value = (255.f - static_cast<float>(iter + 1.5 - std::log(std::log(real * real + img * img)))) / 255.f;
        dest[base_index] = 1.f;
        dest[base_index + 1] = value;
        dest[base_index + 2] = value;
    }
}

// for_each functor
struct apply_mandel {
    typedef thrust::device_vector<float> data_vector;

    apply_mandel(float* vec, int rows, int cols, const int max_iterations) 
        : dest(vec), rows(rows), cols(cols), iterations(max_iterations) {}

    __host__ __device__
    void operator()(int& id) {
        const int col = id % (cols * 3); 
        const int row = id / (cols * 3) ;
        apply_mandel_brot(dest, col, row, rows, cols, iterations);
    }
    
    float* dest;
    const int rows;
    const int cols;
    const int iterations;
};


thrust::host_vector<float> render_mandel_brot_gpu(int rows, int cols, const int iterations) {
	thrust::device_vector<float> d_vec(rows * cols * 3);
    apply_mandel worker(thrust::raw_pointer_cast(&d_vec[0]), rows, cols, iterations);

    thrust::device_vector<int> indices(rows * cols);
    thrust::sequence(indices.begin(), indices.end(), 0, 3);

    thrust::for_each(indices.begin(), indices.end(), worker);

    thrust::host_vector<float> h_vec = d_vec;
    return h_vec;
}

std::vector<float> render_mandel_brot_cpu(int rows, int cols, const int iterations) {
    std::vector<float> vec(rows * cols * 3);
    apply_mandel worker(&vec[0], rows, cols, iterations);

    std::vector<int> indices(rows * cols);
    thrust::sequence(indices.begin(), indices.end(), 0, 3);

    std::for_each(indices.begin(), indices.end(), worker);

    return vec;
}

int main()
{
    typedef std::chrono::high_resolution_clock hr_clock;
    typedef std::chrono::milliseconds ms;

    char mode = ' ';
    int iterations = 100;
    int rows = 4000;
    int cols = rows;

    std::cin >> mode;
    std::cin >> iterations;
    std::cin >> rows;

    cols = rows;

    cv::Mat img;

    if(mode == 'g') {
        std::cout << "running on GPU" << std::endl;
        auto begin = hr_clock::now();
        thrust::host_vector<float> data = render_mandel_brot_gpu(rows, cols, iterations);
        auto end = hr_clock::now();
        std::cout << "GPU took " << std::chrono::duration_cast<ms>(end - begin).count() << std::endl;

        img = copy_to_Mat(data, rows, cols);
    }
    else {
        std::cout << "running on CPU" << std::endl;
        auto begin = hr_clock::now();
        std::vector<float> data = render_mandel_brot_cpu(rows, cols, iterations);
        auto end = hr_clock::now();
        std::cout << "CPU took " << std::chrono::duration_cast<ms>(end - begin).count() << std::endl;
        
        img = copy_to_Mat(data, rows, cols);
    }

	cv::namedWindow("mega", 0); // 0 means resizeable
	cv::imshow("mega", img);
	cv::waitKey(0);

    std::cout << "Writing to file ..." << std::endl;
    cv::Mat transformed = transform_to_uchar(img);
    cv::imwrite("mandelbrot.jpeg", transformed);

    return 0;
}
